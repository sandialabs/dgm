#include "hip/hip_runtime.h"
/** \file matmult.cu
    \brief Cuda kernel for arbitrary (non-square) 
           matrix multiplication.  This implementation
           is single precision (double).
    \author K. Noel Belcourt

    Compile this file with: 
      nvcc -arch sm_35 -I /home/kbelco/local/boost/include matmult.cu 
      -L /home/kbelco/local/boost/lib -l boost_system -l boost_chrono
**/

#include <algorithm>
#include <cassert>
#include <iostream>
#include <iostream>
#include <vector>

#include <boost/chrono.hpp>
#include <boost/chrono/duration.hpp>
#include <boost/chrono/system_clocks.hpp>

using std::ceil;
using std::copy;
using std::cout;
using std::endl;
using std::ostream_iterator;
using std::vector;

using namespace boost::chrono;
using boost::chrono::steady_clock;
using boost::chrono::duration;
using boost::chrono::nanoseconds;
using boost::chrono::milliseconds;

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__ void vector_multiply(double *odata, const double *idata
  , const size_t length)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
    size_t j = (y + i) * width + x;
    if (j < length) 
      odata[j] *= idata[j];
  }
}

__global__ void transpose(double *a, const double *b
  , const size_t n, const size_t m)
{
  // a[m,n] = b[n,m]
  __shared__ double tile[TILE_DIM][TILE_DIM+1];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int col = bx * TILE_DIM + tx;
  int row = by * TILE_DIM + ty;

  for (int j=0; j<TILE_DIM; j+=BLOCK_ROWS) {
    if (row < n && col < m) {
      size_t i = (row + j) * m + col;
      tile[ty+j][tx] = b[i];
    }
  }

  col = by * TILE_DIM + tx;
  row = bx * TILE_DIM + ty;
  __syncthreads();

  for (int j=0; j<TILE_DIM; j+=BLOCK_ROWS) {
    if (row < m && col < n) {
      size_t i = (row + j) * n + col;
      a[i] = tile[tx][ty+j];
    }
  }
}

__global__ void matmult(const double* a, const double* b, double* c
  , size_t width_c, size_t width_a, size_t height_c) 
{
  __shared__ double m[TILE_DIM][TILE_DIM];
  __shared__ double n[TILE_DIM][TILE_DIM];

  size_t bx = blockIdx.x;
  size_t by = blockIdx.y;
  size_t tx = threadIdx.x; 
  size_t ty = threadIdx.y;
  size_t row = by * TILE_DIM + ty;
  size_t col = bx * TILE_DIM + tx;

  double v = 0;
  size_t nb = width_a <= TILE_DIM ? 1 : ceil(double(width_a)/TILE_DIM);
  for (size_t i=0; i<nb; ++i) {
    // zero entry in shared memory
    m[ty][tx] = 0;
    n[ty][tx] = 0;
    if (row < height_c && tx < width_a) {
      // load shared memory from a 
      m[ty][tx] = a[row * width_a + i * TILE_DIM + tx];
    }
    if (i * TILE_DIM + ty < width_a) {
      // load shared memory from b
      n[ty][tx] = b[col + width_c * (i * TILE_DIM + ty)];
    }
    __syncthreads();
    for (int j=0; j<TILE_DIM; ++j) {
      v += m[ty][j] * n[j][tx];
    }
    __syncthreads();
  }
  // store c
  if (row < height_c && col < width_c) {
    c[row * width_c + col] = v;
  }
}

void vector_multiply() {
  // a[n] *= b[n]
  size_t n = 17293154;
  size_t m = ceil(sqrt(n));
  // compute block size
  size_t bs = m <= TILE_DIM ? 1 : ceil(double(m)/TILE_DIM);
  // treat linear vectors as 2d arrays
  cout << "a[" << n << "] *= b[" << n << "]" << endl;
  double* dev_a = 0, *dev_b = 0;
  vector<double> a(n, -2), b(n, -3);

  size_t s = sizeof(double);
  hipError_t err = hipMalloc(&dev_a, s*n);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMalloc(&dev_b, s*n);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_a, &a[0], s*n, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_b, &b[0], s*n, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // 2d blocking for matrices
  dim3 blocks_per_grid(bs, bs);
  dim3 threads_per_block(TILE_DIM, BLOCK_ROWS);

  // very simple sanity check that arrays are big enough
  assert(n <= threads_per_block.x * 4 * threads_per_block.y *
    blocks_per_grid.x * blocks_per_grid.y);

  // time launch and wait for completion before ending timing
  steady_clock::time_point start = steady_clock::now();
  vector_multiply<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, n);
  hipDeviceSynchronize();
  steady_clock::time_point end = steady_clock::now();
  nanoseconds elapsed = end - start;

  // results
  cout << "Vector Multiply Elapsed: " << elapsed.count() << " ns" << endl;
  cout << "  BandWidth: " << s*2*n/elapsed.count() << " GB/s" << endl;
  cout << "  Floating Point Performance: " << n/elapsed.count() << " GFlop/s" << endl;

  // pull data back from device
  err = hipMemcpy(&a[0], dev_a, s*n, hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // same answer at beginning, end and middle
#if 1
  for (size_t i=0; i<n; ++i) {
    if (6 != a[i]) {
      cout << i << " " << a[i] << endl;
    }
  }
#else
  cout << "result a: ";
  cout << "front = " << a[0] << " middle = " << a[a.size()/2] << " back = " << a.back() << endl;
  copy(a.begin(), a.end(), ostream_iterator<double>(cout, " "));
  cout << endl;
#endif
}

void matrix_transpose() {
  // matrix dimensions
  // a[m,n] = b[n,m]
  size_t n = 1724;
  size_t m = n-2;
  cout << "a[" << m << "," << n << "] = b[" << n << "," << m << "]" << endl;
  // compute block size in x (# columns in c) and y (# rows in c)
  int bs_n = n <= TILE_DIM ? 1 : ceil(double(n)/TILE_DIM);
  int bs_m = m <= TILE_DIM ? 1 : ceil(double(m)/TILE_DIM);
  double* dev_a = 0, *dev_b = 0;
  vector<double> a(m*n, -2), b(n*m, -3);
  for (int i=0; i<m*n; ++i) b[i] = i;

  size_t s = sizeof(double);
  hipError_t err = hipMalloc(&dev_a, s*m*n);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMalloc(&dev_b, s*n*m);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_a, &a[0], s*m*n, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_b, &b[0], s*n*m, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // 2d blocking for matrices
  dim3 blocks_per_grid(bs_m, bs_n);
  dim3 threads_per_block(TILE_DIM, BLOCK_ROWS);

  // very simple sanity check that arrays are big enough
  assert(n <= threads_per_block.x * threads_per_block.y *
    blocks_per_grid.x * blocks_per_grid.y);

  // time launch and wait for completion before ending timing
  steady_clock::time_point start = steady_clock::now();
  transpose<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, n, m);
  hipDeviceSynchronize();
  steady_clock::time_point end = steady_clock::now();
  nanoseconds elapsed = end - start;

  // results
  cout << "Matrix Transpose Elapsed: " << elapsed.count() << " ns" << endl;
  cout << "  BandWidth: " << s*2*n*m/elapsed.count() << " GB/s" << endl;

  // pull data back from device
  err = hipMemcpy(&a[0], dev_a, s*n*m, hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // check results
#if 1
  // a[m,n] = b[n,m]
  for (size_t i=0,k=0; i<n; ++i) {
    for (size_t j=0; j<m; ++j, ++k) {
      if (k != a[j*n+i]) {
        cout << " index: " << k << " != " << a[j*n+i] << endl;
      }
    }
  }
#else
  cout << "result a:\n";
  vector<double>::iterator it = a.begin();
  for (size_t i=0; i<m; ++i, it+=n) {
    copy(it, it+n, ostream_iterator<double>(cout, " "));
    cout << endl;
  }
#endif
}

// 260 Gflops if large enough matrices
//   e.g. NxN matrices with N = 9^3 on Curie.
// This implementation supports arbitrary matrix dimensions.
// Round up matrix sizes as necessary so there's no if tests in kernel.
void matrix_multiply() {
  // matrix dimensions
  // c[n,r] = a[n,m] * b[m,r]
  size_t n = 729;
  size_t m = 727;
  size_t r = 731;
  cout << "c[" << n << "," << r << "] = a[" << n << "," << m 
    << "] * b[" << m << "," << r << "]" << endl;
  // compute block size in x (# columns in c) and y (# rows in c)
  size_t bs_r = r <= TILE_DIM ? 1 : ceil(double(r)/TILE_DIM);
  size_t bs_n = n <= TILE_DIM ? 1 : ceil(double(n)/TILE_DIM);
  size_t bs_m = m <= TILE_DIM ? 1 : ceil(double(m)/TILE_DIM);
#if 0
  // handle non-square matrices
  int dn = bs_n * TILE_DIM;
  int dr = bs_r * TILE_DIM;
  int dm = bs_m * TILE_DIM;
#endif
  // number of flops
  size_t n3 = n*r*m;
  double* dev_a = 0, *dev_b = 0, *dev_c = 0;
  vector<double> a(n * m, -1), b(m * r, -2), c(n * r, -5);

#if 0
  a[0] = 1;
  a[1] = 2;
  a[2] = 3;
  a[3] = 4;
  a[4] = 5;
  a[5] = 6;
  a[6] = 7;
  a[7] = 8;
  a[8] = 9;
  a[9] = 10;
  a[10] = 11;
  a[11] = 12;
  a[12] = 2;
  a[13] = 3;
  a[14] = 4;
  a[15] = 4;
  a[16] = 3;
  a[17] = 2;
 
  b[0] = 1;
  b[1] = 2;
  b[2] = 3;
  b[3] = 4;
  b[4] = 5;
  b[5] = 6;
#endif

  size_t s = sizeof(double);
  hipError_t err = hipMalloc(&dev_a, n*m*s);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMalloc(&dev_b, m*r*s);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMalloc(&dev_c, n*r*s);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_a, &a[0], n*m*s, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_b, &b[0], m*r*s, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();
  err = hipMemcpy(dev_c, &c[0], n*r*s, hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // 2d blocking for matrices
  dim3 threads_per_block(TILE_DIM, TILE_DIM);
  dim3 blocks_per_grid(bs_r, bs_n);

  // very simple sanity check that arrays are big enough
  assert(n <= threads_per_block.x * threads_per_block.y *
    blocks_per_grid.x * blocks_per_grid.y);

  // time launch and wait for completion before ending timing
  steady_clock::time_point start = steady_clock::now();
  matmult<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, dev_c, r, m, n);
  hipDeviceSynchronize();
  steady_clock::time_point end = steady_clock::now();
  nanoseconds elapsed = end - start;

  // results
  cout << "Matrix Multiply Elapsed: " << elapsed.count() << " ns" << endl;
  cout << "  BandWidth: " << s*(n*r+n*m+m*r)/elapsed.count() << " GB/s" << endl;
  cout << "  Floating Point Performance: " << 2*n3/elapsed.count() << " GFlop/s" << endl;

  // pull data back from device
  err = hipMemcpy(&c[0], dev_c, n*r*sizeof(double), hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
  hipDeviceSynchronize();

  // same answer at beginning, end and middle
#if 1
  for (size_t i=0; i<n*r; ++i) {
    if (2*m != c[i]) {
      cout << i << " ";
    }
  }
#else
  cout << "result c: ";
  copy(c.begin(), c.end(), ostream_iterator<double>(cout, " "));
  cout << endl;
#endif
}

int main(int, char **) {
  matrix_transpose();
  cout << endl;
  vector_multiply();
  cout << endl;
  matrix_multiply();
  cout << endl;
  return 0;
}
