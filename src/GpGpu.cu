#include "hip/hip_runtime.h"
/** \file GpGpu.hpp
    \brief Gpu utilities like those in Utils.hpp
    \author K. Noel Belcourt
    \author Pedro Bello-Maldonado

    This provides BLAS and LAPACK interfaces for GpGpu
*/

#include <algorithm>
#include <cassert>
#include <fstream>
#include <iostream>
#include <iterator>
#include <ostream>
#include <stdexcept>
#include <string>
#include <vector>

#include "GpGpu.hpp"

#include <boost/lexical_cast.hpp>
#include <boost/numeric/conversion/cast.hpp>

using std::ceil;
using std::copy;
using std::cout;
using std::endl;
using std::distance;
using std::ostream_iterator;
using std::size_t;
using std::string;
using std::swap;
using std::runtime_error;
using std::vector;

using boost::lexical_cast;
using boost::numeric_cast;

const int TILE_DIM = 32;  // TILE_DIM^2 is number of threads in block

#ifdef DGM_USE_CUDA
namespace DGM {

namespace GpGpu {

  const int BLOCK_ROWS = 8;

  // inplace vector multiply
  __global__ void vec_mult(double* a, const double* b,
    const int length)
  {
    a += blockIdx.z * length;
    for (int j=0; j<TILE_DIM; j+=BLOCK_ROWS) {
      int index = ((blockIdx.y * TILE_DIM + threadIdx.y) + j) *
                  (gridDim.x * TILE_DIM)
        + (blockIdx.x * TILE_DIM + threadIdx.x);
      if (index < length) a[index] *= b[index];
    }
  }

  // Theoretical bandwidth on K20X (c.f. http://www.nvidia.com/content/
  // tesla/pdf/nvidia-tesla-kepler-family-datasheet.pdf). Note that
  // this is not my algorithm, though I did modify it to handle
  // rectangular matrices.  Mark Harris of Nvidia seems to be the author
  // of the square transpose implementaiton (c.f. http://devblogs.nvidia.com/
  // parallelforall/efficient-matrix-transpose-cuda-cc/
  __global__ void transpose(double *a, const double *b,
    const int n, const int m)
  {
    __shared__ double tile[TILE_DIM][TILE_DIM+1];

    // a[m,n] = b[n,m]
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = bx * TILE_DIM + tx;
    int row = by * TILE_DIM + ty;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      // load b into column in the cache
      if (row < n && col < m) {
        int i = (row + j) * m + col;
        tile[ty + j][tx] = b[i];
      }
    }

    // transpose block offsets
    col = by * TILE_DIM + tx;
    row = bx * TILE_DIM + ty;
    __syncthreads();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      // store a from a row in the cache
      if (row < m && col < n) {
        int i = (row + j) * n + col;
        a[i] = tile[tx][ty + j];
      }
    }
  }

  // compute c[height_c, width_c] = a[height_c, width_a] * b[width_a, width_c]
  __device__ void device_matmult(const double *a, const double *b, double *c,
    const int width_c, const int width_a, const int height_c)
  {
    __shared__ double m[TILE_DIM][TILE_DIM];
    __shared__ double n[TILE_DIM][TILE_DIM];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_DIM + ty;
    int col = bx * TILE_DIM + tx;

    double v = 0;
    int nb = width_a <= TILE_DIM ? 1 : ceil(double(width_a)/TILE_DIM);
    for (int i=0; i<nb; ++i) {
      // zero shared memory
      m[ty][tx] = 0;
      n[ty][tx] = 0;
      // skip load of a if outside matrix size
      if (row < height_c && tx < width_a) {
        m[ty][tx] = a[row * width_a + i * TILE_DIM + tx];
      }
      // skip load of b if outside matrix size
      if (i * TILE_DIM + ty < width_a) {
        n[ty][tx] = b[col + (i * TILE_DIM + ty) * width_c];
      }
      __syncthreads();
      for (int j=0; j<TILE_DIM; ++j) {
        v += m[ty][j] * n[j][tx];
      }
      __syncthreads();
    }
    if (row < height_c && col < width_c) {
      c[row * width_c + col] = v;
    }
  }

  // compute c[height_c, width_c] = a[height_c, width_a] * b[width_a, width_c]
  __global__ void matmult_load_a_once(double *a, double *b, double *c,
    const int width_c, const int width_a, const int height_c,
    const int inc)
  {
    b += blockIdx.z * inc;
    c += blockIdx.z * inc;
    device_matmult(a, b, c, width_c, width_a, height_c);
  }

  // compute c[height_c, width_c] = a[height_c, width_a] * b[width_a, width_c]
  __global__ void matmult_load_b_once(double *a, double *b, double *c,
    const int width_c, const int width_a, const int height_c,
    const int a_inc, const int c_inc)
  {
    a += blockIdx.z * a_inc;
    c += blockIdx.z * c_inc;
    device_matmult(a, b, c, width_c, width_a, height_c);
  }

  // and calls the real matmult for each submatrix
  // compute c[nm, qb * qc] = a[nm, qb] * b[qa, qb * qc]
  __global__ void submatrix_matmult(double *a, double *b,
    double *c, const int q, const int nm,
    const int b_inc, const int c_inc)
  {
    // each element and submatrix runs in its own block
    b += (blockIdx.z / nm) * b_inc + (blockIdx.z % nm) * q * q;
    c += (blockIdx.z / nm) * c_inc + (blockIdx.z % nm) * nm * q;
    // submatrix sizes of Ha[nm, qc] = Bb[nm, qb] * H[qb, qc]
    device_matmult(a, b, c, q, q, nm);
  }

struct hex_kernels_impl {
  double *wJ;        // weighted Jacobian
  double *B, *BT;  // basis
  double *u;
  double *Ha;  // temporary

  int ne;  // number of elements
  int nm;  // number of modes
  int q, inc;
  int bs_nm, bs_nm2, bs_qbqc, bs_qa, bs_qb, bs_qc;

  hex_kernels_impl() : wJ(0), B(0), BT(0),
    u(0), Ha(0), ne(0), nm(0), q(0), inc(0),
    bs_nm(0), bs_nm2(0), bs_qbqc(0), bs_qa(0), bs_qb(0), bs_qc(0)
  {}

  ~hex_kernels_impl() {
    hipFree(Ha);
    hipFree(u);
    hipFree(BT);
    hipFree(B);
    hipFree(wJ);
    hipDeviceSynchronize();
    Ha = 0;
    u = 0;
    BT = 0;
    B = 0;
    wJ = 0;
  }

  void transpose_B() {
    // setup for transpose: BcT[qc, nm] = Bc[nm, qc];
    // fix to use tiled implementation
    dim3 blocks_per_grid(bs_qc, bs_nm);
    dim3 threads_per_block(TILE_DIM, BLOCK_ROWS);
    transpose<<<blocks_per_grid, threads_per_block>>>(BT, B, nm, q);
    hipDeviceSynchronize();
  }

  // copy from device to host and write to cout
  void output_results(double* it, double* end, string field_name)
  {
    // although the iterators are pointers into gpu memory,
    // distance doesn't dereference them so safe to use on
    // memory that's not locally accessable.
    int s = sizeof(double);
    int d = distance(it, end);
    // cpu memory to hold result
    vector<double> v(d);
    // copy field from device to host memory
    hipError_t err = hipMemcpy(&v[0], it, s * d, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
      throw runtime_error("Error in hex_kernels::u"
        " hipMemcpy failed to copy into device memory for u.");
    hipDeviceSynchronize();
    // output array
    cout << "host " << field_name << ": ";
    copy(v.begin(), v.end(), ostream_iterator<double>(cout, " "));
    cout << endl;
  }

  // should eventually look like Hex::inner_product
  void inner_product() {
    // launch dimensions (z dimension is number of elements)
    dim3 blocks_per_grid(bs_qbqc, bs_qa, ne);
    dim3 threads_per_block(TILE_DIM, BLOCK_ROWS);

    // u *= wJ
    vec_mult<<<blocks_per_grid, threads_per_block>>>(u, wJ, inc);
    hipDeviceSynchronize();

    // u[nm, qb*qc] = B[nm, qa] * u[qa, qb*qc]
    blocks_per_grid = dim3(bs_qbqc, bs_nm, ne);
    threads_per_block = dim3(TILE_DIM, TILE_DIM);
    matmult_load_a_once<<<blocks_per_grid, threads_per_block>>>(
      B, u, u, q*q, q, nm, inc);

    // compute Ha[nm, qb * qc] = B[nm, qb] * u[qa, qb * qc]
    blocks_per_grid = dim3(bs_qc, bs_nm, ne*nm);
    submatrix_matmult<<<blocks_per_grid, threads_per_block>>>(B, u,
      Ha, q, nm, inc, inc);
    hipDeviceSynchronize();

    // u[nm2, nm] = Ha[nm2, qc] * BT[qc, nm]
    int nm2 = nm * nm;
    blocks_per_grid = dim3(bs_nm, bs_nm2, ne);
    matmult_load_b_once<<<blocks_per_grid, threads_per_block>>>(
      Ha, BT, u, nm, q, nm2, inc, nm * nm2);
    hipDeviceSynchronize();
  }
};

hex_kernels::hex_kernels() : impl(0)
{
  impl = new hex_kernels_impl;
}

hex_kernels::~hex_kernels() {
  hipDeviceReset();
  delete impl;
  impl = 0;
}

void hex_kernels::initialize(size_t ne, size_t nmodes, size_t qa
  , size_t qb, size_t qc)
{
  impl->ne = ne;
  impl->q = qa;
  impl->nm = nmodes;
  // size of one element in double precision words used by
  // cuda to increment pointers for threads and blocks
  int inc = impl->inc = qa * qb * qc;
  // int td2 = TILE_DIM * TILE_DIM;
  // overallocate for each element to avoid memory bank conflicts
  // impl->inc = inc = ceil(double(inc)/td2) * td2;
  // compute block sizes, ensure always at least one
  int bs_nm = impl->bs_nm = nmodes <= TILE_DIM ? 1 : ceil(double(nmodes)/TILE_DIM);
  impl->bs_nm2 = nmodes*nmodes <= TILE_DIM ? 1 : ceil(double(nmodes*nmodes)/TILE_DIM);
  int bs_qbqc = impl->bs_qbqc = qa*qa <= TILE_DIM ? 1 : ceil(double(qa*qa)/TILE_DIM);
  int bs_qa = impl->bs_qa = qa <= TILE_DIM ? 1 : ceil(double(qa)/TILE_DIM);
  int bs_qb = impl->bs_qb = qb <= TILE_DIM ? 1 : ceil(double(qb)/TILE_DIM);
  int bs_qc = impl->bs_qc = qc <= TILE_DIM ? 1 : ceil(double(qc)/TILE_DIM);
  // allocate cuda memory for u[qa, qb*qc]
  int s = sizeof(double);
  int count = ne * s * inc;
  hipError_t err = hipMalloc(&impl->u, count);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::init"
      " unable to allocate device memory for U err = " +
      lexical_cast<string>(err) + ".");
  // allocate temporary Ha
  err = hipMalloc(&impl->Ha, count);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::init"
      " unable to allocate device memory for Ha.");
  // allocate weighted Jacobian wJ[qa*qb*qc]
  count = s * inc;
  err = hipMalloc(&impl->wJ, count);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::init"
      " unable to allocate device memory for wJ.");
  // B[nm,qa]
  count = s * nmodes * qa;
  err = hipMalloc(&impl->B, count);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::init"
      " unable to allocate device memory for B.");
  // Bc transpose [qc,nm]
  err = hipMalloc(&impl->BT, count);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::init"
      " unable to allocate device memory for BT.");
  // wait for allocation kernels to finishs
  hipDeviceSynchronize();
}

// set weighted Jacobian
template <>
void hex_kernels::wJ<double*>(double* it, double* end) {
  int s = sizeof(double);
  // length of memory for one element
  int d = distance(it, end);
  // simple check that the sizes match
  int inc = impl->inc;
  assert(d == inc);
  // copy wJ into Cuda memory as 2d array [qa, qb*qc]
  hipError_t err = hipMemcpy(impl->wJ, it, s*inc, hipMemcpyHostToDevice);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::wJ"
      " hipMemcpy failed to copy into device memory for wJ.");
  hipDeviceSynchronize();
}

// set Basis vector
template <>
void hex_kernels::Basis<double*>(double* ba_it, double* bb_it
  , double* bc_it)
{
  int s = sizeof(double);
  int nm = impl->nm;
  int q = impl->q;
  // copy in basis memory B[nm,qa]
  hipError_t err = hipMemcpy(impl->B, ba_it, s * nm * q, hipMemcpyHostToDevice);
  if (err != hipSuccess)
    throw runtime_error("Error in hex_kernels::Basis"
      " hipMemcpy failed to copy into device memory for B, err = " +
      lexical_cast<string>(err) + ".");
  hipDeviceSynchronize();
  // use Cuda to transpose B into BT
  impl->transpose_B();
}

template <>
void hex_kernels::inner_product<double*>(double* it, double* ht) {
  // set U field
  int s = sizeof(double);
  // data in one element
  int inc = impl->inc;
  int ne = impl->ne;
  // copy in field data u[qa, qb*qc] from host to device
  if (hipSuccess != hipMemcpy(impl->u, it, s * ne * inc,
    hipMemcpyHostToDevice))
  {
    throw runtime_error("Error in hex_kernels::u"
      " hipMemcpy failed to copy into device memory for u.");
  }
  hipDeviceSynchronize();
  // invoke gpu kernels
  impl->inner_product();
  // copy out field data from device to host
  int nm = impl->nm;
  int nm3 = nm * nm * nm;
  // device to host copy
  for (int i=0; i<ne; ++i) {
    double *u_it = ht + i * inc;
    if (hipSuccess != hipMemcpy(u_it, impl->u + i * nm3, s * nm3,
      hipMemcpyDeviceToHost))
    {
      throw runtime_error("Error in hex_kernels::u"
        " hipMemcpy failed to copy into host from device memory for u.");
    }
  }
  // wait for all memcpy kernels to finish
  hipDeviceSynchronize();
}

template<>
double dot<double,int>(const size_t n, const double* x, const double* y,
  int nx, int ny)
{
  cout<<"CUDA dot product not implemented"<<endl;
  return 0;
}

template<>
float dot<float,int>(const size_t n, const float* x, const float* y,
  int nx, int ny)
{
  cout<<"CUDA dot product not implemented"<<endl;
  return 0;
}

} // namespace GpGpu
} // namespace DGM

#endif  // defined DGM_USE_CUDA

// Kokkos specific code
#ifdef DGM_USE_KOKKOS

#define KOKKOS_HAVE_CUDA
#include "Kokkos_Core.hpp"
#include "Kokkos_DualView.hpp"
#include "Cuda/Kokkos_Cuda_Parallel.hpp"

using Kokkos::Cuda;
using Kokkos::DualView;
using Kokkos::LayoutLeft;
using Kokkos::LayoutRight;
using Kokkos::RangePolicy;
using Kokkos::TeamPolicy;
using Kokkos::TeamThreadRange;
using Kokkos::ThreadVectorRange;
using Kokkos::View;

using Kokkos::finalize;
using Kokkos::initialize;
using Kokkos::fence;
using Kokkos::parallel_for;

namespace {
  // track number of times Kokkos is initialized
  static int instances = 0;

  // execution space
  typedef Cuda Space;

  // layout
  typedef LayoutLeft Layout;

  // views
  typedef View<double****, Layout, Space> ArrayView4D;
  typedef View<double***, Layout, Space> ArrayView3D;
  typedef View<double**, Layout, Space> ArrayView2D;
  typedef DualView<double****, Layout, Space> ArrayDualView4D;
  typedef DualView<double***, Layout, Space> ArrayDualView3D;
  typedef DualView<double**, Layout, Space> ArrayDualView2D;

  // policy
  typedef TeamPolicy<Space> TeamPolicyExec;
  typedef TeamPolicy<Space>::member_type MemberTypeExec;
}

namespace DGM {
namespace GpGpu {

  struct hex_kernels_impl {
    // u[num_elem, quad_a, quad_b, quad_c]
    ArrayView4D u_dev;
    ArrayView4D::HostMirror u_hst;
    ArrayDualView4D u;

    // H[num_elem, num_modes, quad_b, quad_c]
    ArrayView4D H_dev;
    ArrayView4D::HostMirror H_hst;
    ArrayDualView4D H;

    // Ha[num_elem, num_modes, num_modes, quad_c]
    ArrayView4D Ha_dev;
    ArrayView4D::HostMirror Ha_hst;
    ArrayDualView4D Ha;

    // wJ[quad_a, quad_b, quad_c]
    ArrayView3D wJ_dev;
    ArrayView3D::HostMirror wJ_hst;
    ArrayDualView3D wJ;

    // Ba[num_elem, quad_a], Bb[num_elem, quad_b], Bc[num_elem, quad_c]
    ArrayView2D Ba_dev, Bb_dev, Bc_dev;
    ArrayView2D::HostMirror Ba_hst, Bb_hst, Bc_hst;
    ArrayDualView2D Ba, Bb, Bc;

    int num_elem;
    int num_modes;
    int quad_a, quad_b, quad_c;
    int length;

    hex_kernels_impl() : num_elem(0), num_modes(0), quad_a(0)
      , quad_b(0), quad_c(0), length(0) {
    }

    void inner_product() {
      int num_elem = this->num_elem;
      int num_modes = this->num_modes;
      int quad_a = this->quad_a;
      int quad_b = this->quad_b;
      int quad_c = this->quad_c;

      int num_threads = 128;

      // For each element in num_elem, compute
      // u[num_elem, quad_a, quad_b, quad_c] *= wJ[quad_a, quad_b, quad_c]
      double* u_ptr = u_dev.data();
      double* wJ_ptr = wJ_dev.data();

      parallel_for(TeamPolicyExec(num_elem, num_threads), 
        [u_ptr, wJ_ptr, num_elem, quad_a, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        const int n_e = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, quad_a * quad_b * quad_c), 
          [=] (const int idx)
        {
          u_ptr[n_e * (quad_a * quad_b * quad_c) + idx] *= wJ_ptr[idx];
        });
      });

      // For each element in num_elem, compute
      // H[num_elem, num_modes, quad_b * quad_c] = 
      // Ba[num_modes, quad_a] * u[num_elem, quad_a, quad_b * quad_c]
      int block_x = 16;
      int block_y = 16;
      double* H_ptr = H_dev.data();
      double* Ba_ptr = Ba_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [H_ptr, Ba_ptr, u_ptr, num_elem, num_modes, quad_a, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        int n_e = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, num_modes), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, quad_b * quad_c), 
            [=] (const int j)
          {
            int n = num_modes;
            int m = quad_b * quad_c;
            int p = quad_a;
            double sum = 0.0;
            double* A = Ba_ptr;
            double* B = u_ptr + n_e * (p * m);
            double* C = H_ptr + n_e * (n * m);

            for (int k=0; k<p; ++k) {
              sum += A[i * p + k] * B[k * m + j];
            }
            C[i * m + j] = sum;
          });
        });
      });

      // For each element in num_elem, compute
      // Ha[num_elem, num_modes, num_modes, quad_c] = 
      // Bb[num_modes, quad_b] * H[num_elem, num_modes, quad_b, quad_c]
      double* Bb_ptr = Bb_dev.data();
      double* Ha_ptr = Ha_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [Ha_ptr, Bb_ptr, H_ptr, num_elem, num_modes, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        int n_e = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, num_modes), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, quad_c), 
            [=] (const int j)
          {
            int n = num_modes;
            int m = quad_c;
            int p = quad_b;
            double sum = 0.0;
            double* A = Bb_ptr;

            for (int n_m = 0; n_m < num_modes; ++n_m) {
              double* B = H_ptr + n_e * (n * m * p) + n_m * (p * m);
              double* C = Ha_ptr + n_e * (n * n * m) + n_m * (n * m);

              sum = 0.0;
              for (int k=0; k<p; ++k) {
                sum += A[i * p + k] * B[k * m + j];
              }
              C[i * m + j] = sum;
            }
          });
        });
      });

      // For each element in num_elem, compute
      // u[num_elem, num_modes, num_modes, num_modes] = 
      // Ha[num_elem, num_modes, num_modes, quad_c] * Bc.T[quad_c, num_modes]
      double* Bc_ptr = Bc_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [u_ptr, Ha_ptr, Bc_ptr, num_elem, num_modes, quad_a, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        int n_e = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, num_modes), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, num_modes), 
            [=] (const int j)
          {
            int n = num_modes;
            int m = num_modes;
            int p = quad_c;
            double sum = 0.0;
            double* B = Bc_ptr;

            for (int n_m = 0; n_m < num_modes; ++n_m) {
              double* A = Ha_ptr + n_e * (n * m * p) + n_m * (n * p);
              double* C = u_ptr + n_e * (quad_a * quad_b * quad_c) + n_m * (n * m);

              sum = 0.0;
              for (int k=0; k<p; ++k) {
                sum += A[i * p + k] * B[j * p + k];
              }
              C[i * m + j] = sum;
            }
          });
        });
      });
    }

    void backward_transform() {
      // HexBlock Data
      int num_elem = this->num_elem;
      int num_modes = this->num_modes;
      int quad_a = this->quad_a;
      int quad_b = this->quad_b;
      int quad_c = this->quad_c;

      // For each element in num_elem, compute
      // H[num_elem, quad_a, num_modes, num_modes] = 
      // Ba.T[quad_a, num_modes] * u[num_elem, num_modes, num_modes, num_modes]
      int block_x = 16;
      int block_y = 16;
      double* u_ptr = u_dev.data();
      double* H_ptr = H_dev.data();
      double* Ba_ptr = Ba_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [H_ptr, Ba_ptr, u_ptr, num_elem, num_modes, quad_a, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        int elem = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, quad_a), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, num_modes * num_modes), 
            [=] (const int j)
          {
            int n = quad_a;
            int m = num_modes * num_modes;
            int p = num_modes;
            double sum = 0.0;
            double* A = Ba_ptr;
            double* B = u_ptr + elem * (quad_a * quad_b * quad_c);
            double* C = H_ptr + elem * (quad_a * num_modes * num_modes);

            for (int k=0; k<p; ++k) {
              sum += A[k * n + i] * B[k * m + j];
            }
            C[i * m + j] = sum;
          });
        });
      });

      // For each element in num_elem and for each quadrature point, compute
      // Ha[num_elem, quad_a, quad_b, num_modes] = 
      // Bb.T[quad_b, num_modes] * H[num_elem, quad_a, num_modes, num_modes]
      double* Bb_ptr = Bb_dev.data();
      double* Ha_ptr = Ha_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [Ha_ptr, Bb_ptr, H_ptr, num_elem, num_modes, quad_a, quad_b] 
        __device__ (const MemberTypeExec& team_member)
      {
        int elem = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, quad_b), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, num_modes), 
            [=] (const int j)
          {
            int n = quad_b;
            int m = num_modes;
            int p = num_modes;
            double sum = 0.0;
            double* A = Bb_ptr;

            for (int q_a = 0; q_a < quad_a; ++q_a) {
              double* B = H_ptr + elem * (quad_a * num_modes * num_modes) 
                        + q_a * (num_modes * num_modes);
              double* C = Ha_ptr + elem * (quad_a * quad_b * num_modes) 
                        + q_a * (quad_b * num_modes);

              sum = 0.0;
              for (int k=0; k<p; ++k) {
                sum += A[k * n + i] * B[k * m + j];
              }
              C[i * m + j] = sum;
            }
          });
        });
      });

      // For each element in num_elem, compute
      // u[num_elem, quad_a, quad_b, quad_c] = 
      // Ha[num_elem, quad_a, quad_b, num_modes] * Bc[num_modes, quad_c]
      double* Bc_ptr = Bc_dev.data();

      parallel_for(TeamPolicyExec(num_elem, block_y, block_x), 
        [u_ptr, Ha_ptr, Bc_ptr, num_elem, num_modes, quad_a, quad_b, quad_c] 
        __device__ (const MemberTypeExec& team_member)
      {
        int elem = team_member.league_rank();

        parallel_for(TeamThreadRange(team_member, quad_a * quad_b), 
          [=] (const int i)
        {
          parallel_for(ThreadVectorRange(team_member, quad_c), 
            [=] (const int j)
          {
            int n = quad_a * quad_b;
            int m = quad_c;
            int p = num_modes;
            double sum = 0.0;
            double* A = Ha_ptr + elem * (quad_a * quad_b * num_modes);
            double* B = Bc_ptr;
            double* C = u_ptr + elem * (quad_a * quad_b * quad_c);

            for (int k=0; k<p; ++k) {
              sum += A[i * p + k] * B[k * m + j];
            }
            C[i * m + j] = sum;
          });
        });
      });
    }
  };

  hex_kernels::hex_kernels() : impl(0) {
    impl = new hex_kernels_impl;
  }

  hex_kernels::~hex_kernels() {
    delete impl;
    impl = 0;
  }

  int hex_kernels::size_grid(int num_elements, int num_threads) {
    return (num_elements + (num_threads - 1)) / num_threads;
  }

  void hex_kernels::initialize(size_t num_elem, size_t num_modes
    , size_t quad_a, size_t quad_b, size_t quad_c)
  {
    impl->num_elem = num_elem;
    impl->num_modes = num_modes;
    impl->quad_a = quad_a;
    impl->quad_b = quad_b;
    impl->quad_c = quad_c;

    // Length must be a power of 2
    impl->length = 1;

    while ((impl->length < quad_c / 2) && (impl->length < TILE_DIM)) {
      impl->length *= 2;
    }
  }

  // Set weighted Jacobian
  template <>
  void hex_kernels::wJ(double* wJ_ptr) {
    int quad_a = impl->quad_a;
    int quad_b = impl->quad_b;
    int quad_c = impl->quad_c;

    impl->wJ_dev = ArrayView3D("wJ_dev", quad_a, quad_b, quad_c);
    impl->wJ_hst = ArrayView3D::HostMirror(wJ_ptr, quad_a, quad_b, quad_c);
    impl->wJ = ArrayDualView3D(impl->wJ_dev, impl->wJ_hst);

    impl->wJ.modify<ArrayDualView3D::host_mirror_space>();
    impl->wJ.sync<ArrayDualView3D::memory_space>();
  }

  template<>
  void hex_kernels::Basis(double* Ba_ptr, double* Bb_ptr, double* Bc_ptr) {
    int quad_a = impl->quad_a;
    int quad_b = impl->quad_b;
    int quad_c = impl->quad_c;
    int num_modes = impl->num_modes;

    impl->Ba_dev = ArrayView2D("Ba_dev", num_modes, quad_a);
    impl->Ba_hst = ArrayView2D::HostMirror(Ba_ptr, num_modes, quad_a);
    impl->Ba = ArrayDualView2D(impl->Ba_dev, impl->Ba_hst);

    impl->Bb_dev = ArrayView2D("Bb_dev", num_modes, quad_b);
    impl->Bb_hst = ArrayView2D::HostMirror(Bb_ptr, num_modes, quad_b);
    impl->Bb = ArrayDualView2D(impl->Bb_dev, impl->Bb_hst);

    impl->Bc_dev = ArrayView2D("Bc_dev", num_modes, quad_c);
    impl->Bc_hst = ArrayView2D::HostMirror(Bc_ptr, num_modes, quad_c);
    impl->Bc = ArrayDualView2D(impl->Bc_dev, impl->Bc_hst);

    impl->Ba.modify<ArrayDualView2D::host_mirror_space>();
    impl->Ba.sync<ArrayDualView2D::memory_space>();

    impl->Bb.modify<ArrayDualView2D::host_mirror_space>();
    impl->Bb.sync<ArrayDualView2D::memory_space>();

    impl->Bc.modify<ArrayDualView2D::host_mirror_space>();
    impl->Bc.sync<ArrayDualView2D::memory_space>();
  }

  template<>
  void hex_kernels::inner_product(double* u_ptr) {
    int quad_a = impl->quad_a;
    int quad_b = impl->quad_b;
    int quad_c = impl->quad_c;
    int num_elem = impl->num_elem;
    int num_modes = impl->num_modes;

    impl->u_dev = ArrayView4D("u_dev", num_elem, quad_a, quad_b, quad_c);
    impl->u_hst = ArrayView4D::HostMirror(u_ptr, num_elem, quad_a, quad_b, quad_c);
    impl->u = ArrayDualView4D(impl->u_dev, impl->u_hst);

    impl->H_dev = ArrayView4D("H_dev", num_elem, num_modes, quad_b, quad_c);
    impl->H_hst = ArrayView4D::HostMirror("H_hst", num_elem, num_modes, quad_b, quad_c);
    impl->H = ArrayDualView4D(impl->H_dev, impl->H_hst);

    impl->Ha_dev = ArrayView4D("Ha_dev", num_elem, num_modes, num_modes, quad_c);
    impl->Ha_hst = ArrayView4D::HostMirror("Ha_hst", num_elem, num_modes, num_modes, quad_c);
    impl->Ha = ArrayDualView4D(impl->Ha_dev, impl->Ha_hst);

    impl->u.modify<ArrayDualView4D::host_mirror_space>();
    impl->u.sync<ArrayDualView4D::memory_space>();

    impl->inner_product();

    impl->u.modify<ArrayDualView4D::memory_space>();
    impl->u.sync<ArrayDualView4D::host_mirror_space>();

    fence();
  }

  void hex_kernels::backward_transform(double* u_ptr) {
    // Variables
    int quad_a = impl->quad_a;
    int quad_b = impl->quad_b;
    int quad_c = impl->quad_c;
    int num_elem = impl->num_elem;
    int num_modes = impl->num_modes;

    // Instantiate views
    impl->u_dev = ArrayView4D("u_dev", num_elem, quad_a, quad_b, quad_c);
    impl->u_hst = ArrayView4D::HostMirror(u_ptr, num_elem, quad_a, quad_b, quad_c);
    impl->u = ArrayDualView4D(impl->u_dev, impl->u_hst);

    impl->H_dev = ArrayView4D("H_dev", num_elem, quad_a, num_modes, num_modes);
    impl->H_hst = ArrayView4D::HostMirror("H_hst", num_elem, quad_a, num_modes, num_modes);
    impl->H = ArrayDualView4D(impl->H_dev, impl->H_hst);

    impl->Ha_dev = ArrayView4D("Ha_dev", num_elem, quad_a, quad_b, num_modes);
    impl->Ha_hst = ArrayView4D::HostMirror("Ha_hst", num_elem, quad_a, quad_b, num_modes);
    impl->Ha = ArrayDualView4D(impl->Ha_dev, impl->Ha_hst);

    // Synchronize host and device
    impl->u.modify<ArrayDualView4D::host_mirror_space>();
    impl->u.sync<ArrayDualView4D::memory_space>();

    // Invoke kernels
    impl->backward_transform();

    // Synchronize host and device
    impl->u.modify<ArrayDualView4D::memory_space>();
    impl->u.sync<ArrayDualView4D::host_mirror_space>();
  }

  initialize_gpu::initialize_gpu() {
    if (0 == instances) {
      initialize();
    }
    ++instances;
  }

  initialize_gpu::~initialize_gpu() {
    if (0 == --instances) {
      finalize();
    }
  }

template <typename T, typename View>
struct dot_ {
  typedef T value_type;

  View x, y;
  dot_(View x_, View y_) : x(x_), y(y_) {}

  KOKKOS_INLINE_FUNCTION
  void operator()(int i, T& result) const {
    result += x(i) * y(i);
  }
};

template <typename View, typename T, typename S>
void strided_copy(View v, const int n, const T* x, S nx) {
  if (1 == nx) {
    copy(x, x+n, &v[0]);
  }
  else {
    for (S i=0; i<n; ++i)
      v[i] = x[i*nx];
  }
}

#if 0
// Host, View, Type, Scalar
template <typename H, typename V, typename T, typename S>
T dot_t(const int n, const T* x, const T* y, S nx, S ny) {
  view_t vx("x", n), vy("y", n);

  host_view_t hx = Kokkos::create_mirror_view(vx);
  strided_copy(hx, n, x, nx);
  Kokkos::deep_copy(vx, hx);

  host_view_t hy = Kokkos::create_mirror_view(vy);
  strided_copy(hy, n, y, ny);
  Kokkos::deep_copy(vy, hy);

  T result = 0;
  Kokkos::parallel_reduce(n, dot_<T, view_t>(vx, vy), result);
  return result;
}
#endif

#if 0
template <>
double dot<double, int>(const int n, const double* x, const double* y,
  int nx, int ny) {
#if 0
  if (n != fv[0].size()) {
    // x
    dv[0] = double_view_t("x", n);
    dhv[0] = Kokkos::create_mirror_view(dv[0]);
    // y
    dv[1] = double_view_t("y", n);
    dhv[1] = Kokkos::create_mirror_view(dv[1]);
  }
  // copy data to device
  strided_copy(dhv[0], n, x, nx);
  Kokkos::deep_copy(dv[0], dhv[0]);
  strided_copy(dhv[1], n, y, ny);
  Kokkos::deep_copy(dv[1], dhv[1]);
  // launch kernel
  double result = 0;
  Kokkos::parallel_reduce(n, dot_<double, double_view_t>(dv[0], dv[1]), result);
  return result;
#endif
  return 0;
}

template <>
float dot<float, int>(const int n, const float* x, const float* y,
  int nx, int ny) {
#if 0
  if (n != fv[0].size()) {
    // x
    fv[0] = float_view_t("x", n);
    fhv[0] = Kokkos::create_mirror_view(fv[0]);
    // y
    fv[1] = float_view_t("y", n);
    fhv[1] = Kokkos::create_mirror_view(fv[1]);
  }
  // copy data to device
  strided_copy(fhv[0], n, x, nx);
  Kokkos::deep_copy(fv[0], fhv[0]);
  strided_copy(fhv[1], n, y, ny);
  Kokkos::deep_copy(fv[1], fhv[1]);
  // launch kernel
  float result = 0;
  Kokkos::parallel_reduce(n, dot_<float, float_view_t>(fv[0], fv[1]), result);
  return result;
#endif
  return 0;
}
#endif

template<typename Size, typename Array, typename Stride>
Array dot(Size n, Array x, Array y, Size nx, Size ny)
{
    return Array();
}

}  // namespace GpGpu
}  // namespace DGM

#endif // defined DGM_USE_KOKKOS
